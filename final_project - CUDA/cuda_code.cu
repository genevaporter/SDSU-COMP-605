#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "./my_cuda_header.h"

#define BLOCKSIZE 16

//Kernel that performs Matrix Vector Multiplication
__global__ void MatrixVectorMultiplication(double *matrixA,
                                           double *matrixB,
                                           double *matrixC, 
                                           int i_total, 
                                           int j_total, 
                                           int k_total, 
                                           int NumberofProcessors)
{ 	
    for (int i = 0; i < i_total / NumberofProcessors + 1; i++) 
    {
        for (int k = 0; k < k_total; k++) 
        {
            double sum = 0.0;
            for (int j = 0; j < j_total; j++)
                sum += matrixA[i * j_total + j] * matrixB[j * k_total + k];
		
            matrixC[i * k_total + k] = sum;
        }
    }
    __syncthreads();
}

    
void implement_cuda_code(int mpi_rank, int mpi_processors,
                         int i_total, int j_total, int k_total,
                         double *device_matrixA,
                         double *device_matrixB,
                         double *device_matrixC,
                         double *host_matrixA,
                         double *host_matrixB,
                         double *host_matrixC,
                         int threads_per_block, int number_of_blocks)
{
    hipSetDevice(mpi_rank);

    //Allocating the Memory on the device memory
    hipHostAlloc( (void **)&device_matrixA, 
                   i_total * j_total / mpi_processors * sizeof(double), 
                   hipHostMallocDefault );
    hipHostAlloc( (void **)&device_matrixB, 
                   j_total * k_total*sizeof(double), 
                   hipHostMallocDefault);
    hipHostAlloc( (void **)&device_matrixC, 
                   i_total * k_total / mpi_processors * sizeof(double), 
                   hipHostMallocDefault );

    //Copying the data from host to device
    hipMemcpyAsync( (void *)device_matrixA, 
                     (void *)host_matrixA, 
                     i_total * j_total / mpi_processors * sizeof(double), 
                     hipMemcpyHostToDevice );
    hipMemcpyAsync( (void *)device_matrixB, 
                     (void *)host_matrixB,  
                     j_total * k_total * sizeof(double), 
                     hipMemcpyHostToDevice );
    
    hipSetDevice(mpi_rank);

    //Calling the kernel which performs Matrix Vector Product
    MatrixVectorMultiplication<<<number_of_blocks, threads_per_block>>>(device_matrixA, device_matrixB, device_matrixC, i_total, j_total, k_total, mpi_processors);		

    //Copying the value of patial result vector from device to host
    hipMemcpy( (void *)host_matrixC, 
                (void *)device_matrixC, 
                i_total * k_total / mpi_processors * sizeof(double), 
                hipMemcpyDeviceToHost );
    
}

void free_device_memory(double **device_matrixA, 
                        double **device_matrixB, 
                        double **device_matrixC)
{
    hipFree( *device_matrixA );
    hipFree( *device_matrixB );
    hipFree( *device_matrixC );
}




