#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void rbg_to_grayscale(unsigned char * grayscale_image, unsigned char * rbg_image, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) 
	{
		int grayscale_address = y*width + x;
		int rgb_address = grayscale_offset * 3;

		unsigned char R = rgb_image[rgb_address];
		unsigned char G = rgb_image[rgb_address + 1];
		unsigned char B = rgb_image[rgb_address + 2];

		grayscale_image[grayscale_address] = 0.3f*R + 0.59f*G + 0.11f*B;
	} 
}

int main(int argc, char **argv)
{
	// Import image to convert
	unsigned char *rgb_storage, *grayscale_storage;

	// Find width/height of image (in pixels)
	// const int image_size =  width*height ???
	// const int N = image_size * 3 ???

	//Allocate memory for arrays on host
	hipMalloc((void**) &rgb_storage, image_size);
	checkCudaError("Input image memory not allocated.\n");

	hipMalloc((void**) &grayscale_storage, image_size);
	cudaCheckError("Grayscale image memory not allocated.\n");

	size_t bytes = N*sizeof(int);

	// Allocate memory for arrays on device (copy to device)
	


	// FIll array on host ??
	
	// Copy data from host array to device array

	// Convert image to grayscale (run kernel) (do something on device??)
	int threads_per_block = srttol(argv[1], NULL, 10);
	if ( argc != 2 || threads_per_block == 0 )
	{
		cout << "Invalid/missing argument. Block size will be set to 256." 
			 << endl;
		threads_per_block = 256;
	}
	
	int number_of_blocks = N / threads_per_block + 1;

	NSTimer computation_time = NSTimer("computation_time", false, false);
	computation_time.start();
	
	rgb_to_grayscale
		<< number_of_blocks, threads_per_block>>
		(grayscale_image, rbg_image, width, height);

	comoputation_time.stop();

	cudaDeviceSynchroonize();

	// Copy data from device array to host array??

	hipMemcpy(grayscale_image, 

	// Check data for correctness??


	// Free host memory
	free();
	free();

	// Free device memory
	hipFree(rgb_storage);
	hipFree(grayscale_storage);

	return 0;

}









