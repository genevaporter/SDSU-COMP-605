#include "hip/hip_runtime.h"
# define RBG_CHANNELS 3

__global__ void rbg_to_grayscale(unsigned char * grayscale_image, unsigned char * rbg_image, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) 
    {   
        int grayscale_offset = y*width + x;
        int rgb_offset = grayscale_offset * CHANNELS;

        unsigned char R = rgb_image[rgb_offset];
        unsigned char G = rgb_image[rgb_offset + 1]; 
        unsigned char B = rgb_image[rgb_offset + 2]; 

        grayscale_image[grayscale_offset] = 0.3f*R + 0.59f*G + 0.11f*B;
    }   
}

int main(int argc, char **argv)
{
    // Allocate memory for image

    // Import image to convert

    // Duplicate image for conversion?

    // Find width/height of image (in pixels)
    // N = number of elements to process * 3???

    //Allocate memory for arrays on host
    size_t bytes = N*sizeof(int);
    int *size_x = (int*)malloc(bytes);
    int *size_y = (int*)malloc(bytes);

    // Allocate memory for arrays on device
    int x, y;
    hipMallocManaged(&x, N*sizeof(float));    hipMallocManaged(&y, N*sizeof(float));

    // FIll array on host ??

    // Copy data from host array to device array

    // Convert image to grayscale (run kernel) (do something on device??)
    int threads_per_block = srttol(argv[1], NULL, 10);
    if ( argc != 2 || threads_per_block == 0 )
    {
        cout << "Invalid/missing argument. Block size will be set to 256."
             << endl;
        threads_per_block = 256;
    }

    int number_of_blocks = N / threads_per_block + 1;

    rgb_to_grayscale<< number_of_blocks, threads_per_block>>();
    cudaDeviceSynchroonize();

    // Copy data from device array to host array??

    // Check data for correctness??


    // Free host memory
    free(size_x);
    free(size_y);

    // Free device memory
    hipFree(x);
    hipFree(y);

    return 0;

}
                                                      

